#include "hip/hip_runtime.h"

#include "hipblas.h"

#include <iostream>
#include <iomanip>
#include <cstdlib>

#define IDX2C(i, j, ld) (((j) * (ld)) + (i))

float* getMatrix(const int ldm, const int n);
void printMatrix(const float* matrix, const int ldm, const int n);

int main()
{
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    hipStream_t stream;

    // prepare input matrices
    float* pf_A, * pf_B, * pf_C;
    float* df_A, * df_B, * df_C;
    int M, N, K;
    float alpha, beta;

    M = 4;
    N = 5;
    K = 6;
    alpha = 1.f;
    beta = 1.f;

    // create cuBLAS handle
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed\n";
        return EXIT_FAILURE;
    }

    srand(2021);

    pf_A = getMatrix(K, M);
    pf_B = getMatrix(N, K);
    pf_C = getMatrix(M, N);

    std::cout << "A:\n";
    printMatrix(pf_A, K, M);
    std::cout << "B:\n";
    printMatrix(pf_B, N, K);
    std::cout << "C:\n";
    printMatrix(pf_C, M, N);

    // allocate device memory
    hipMalloc((void**)&df_A, M * K * sizeof(float));
    hipMalloc((void**)&df_B, K * N * sizeof(float));
    hipMalloc((void**)&df_C, M * N * sizeof(float));

    // create stream
    cudaStat = hipStreamCreate(&stream);

    // asynchronously set cublas matrix
    hipblasSetMatrixAsync(M, K, sizeof(*df_A), pf_A, M, df_A, M, stream);
    hipblasSetMatrixAsync(K, N, sizeof(*df_B), pf_B, K, df_B, K, stream);
    hipblasSetMatrixAsync(M, N, sizeof(*df_C), pf_C, M, df_C, M, stream);

    hipblasSetStream(handle, stream);

    // gemm
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        M, N, K,
        &alpha,
        df_A, M,
        df_B, K,
        &beta,
        df_C, M);

    hipblasGetMatrixAsync(M, N, sizeof(*df_C), df_C, M, pf_C, M, stream);

    hipStreamSynchronize(stream);
    std::cout << "C out:\n";
    printMatrix(pf_C, M, N);

    hipblasDestroy(handle);
    hipStreamDestroy(stream);

    hipFree(df_A);
    hipFree(df_B);
    hipFree(df_C);

    free(pf_A);
    free(pf_B);
    free(pf_C);

    return 0;
}

float* getMatrix(const int ldm, const int n)
{
    float* pf_matrix = (float*)malloc(ldm * n * sizeof(float));

    for (int j = 0; j < n; j++)
    {
        for (int i = 0; i < ldm; i++)
        {
            pf_matrix[IDX2C(i, j, ldm)] = (float)rand() / RAND_MAX;
        }
    }

    return pf_matrix;
}

void printMatrix(const float* matrix, const int ldm, const int n)
{
    for (int j = 0; j < n; j++)
    {
        for (int i = 0; i < ldm; i++)
        {
            std::cout << std::fixed << std::setw(8) << std::setprecision(4) << matrix[IDX2C(i, j, ldm)];
        }
        std::cout << '\n';
    }
}


#include <hip/hip_runtime.h>
﻿/*#include "cuda_runtime.h"
#include "device_launch_parameters.h"
#include "cublas_v2.h"

#include <iostream>
#include <iomanip>
#include <cstdlib>

#define IDX2C(i, j, ld) (((j) * (ld)) + (i))

float* getMatrix(const int m, const int ldm);
void printMatrix(const float* matrix, const int m, const int ldm);

int main()
{
    cublasHandle_t handle;

    // prepare input matrices
    float* A, * B, * C;
    int M, N, K;
    float alpha, beta;

    M = 3;
    N = 4;
    K = 7;
    alpha = 1.f;
    beta = 0.f;

    // create cuBLAS handle
    if (cublasCreate(&handle) != CUBLAS_STATUS_SUCCESS)
    {
        std::cout << "CUBLAS initialization failed\n";
        return EXIT_FAILURE;
    }

    srand(2021);

    A = getMatrix(K, M);
    B = getMatrix(N, K);
    C = getMatrix(M, N);

    std::cout << "A:\n";
    printMatrix(A, K, M);
    std::cout << "B:\n";
    printMatrix(B, N, K);
    std::cout << "C:\n";
    printMatrix(C, M, N);

    // gemm
    cublasSgemm(handle, CUBLAS_OP_T, CUBLAS_OP_T,
        M, N, K,
        &alpha,
        A, K,
        B, N,
        &beta,
        C, M);

    cudaDeviceSynchronize();
    std::cout << "C out:\n";
    printMatrix(C, M, N);

    cublasDestroy(handle);

    cudaFree(A);
    cudaFree(B);
    cudaFree(C);

    return 0;
}

float* getMatrix(const int m, const int ldm)
{
    float* pf_matrix = nullptr;
    cudaMallocManaged((void**)&pf_matrix, sizeof(float) * ldm * m);

    for (int j = 0; j < m; j++)
    {
        for (int i = 0; i < ldm; i++)
        {
            pf_matrix[IDX2C(i, j, ldm)] = (float)rand() / RAND_MAX;
        }
    }

    return pf_matrix;
}

void printMatrix(const float* matrix, const int m, const int ldm)
{
    for (int j = 0; j < m; j++)
    {
        for (int i = 0; i < ldm; i++)
        {
            std::cout << std::fixed << std::setw(8) << std::setprecision(4) << matrix[IDX2C(i, j, ldm)];
        }
        std::cout << '\n';
    }
}
*/